#include "hip/hip_runtime.h"
#include "emc.h"


__device__ inline void atomicFloatAdd(float *address, float val)
{
  int i_val = __float_as_int(val);
  int tmp0 = 0;
  int tmp1;

  while( (tmp1 = atomicCAS((int *)address, tmp0, i_val)) != tmp0)
    {
      tmp0 = tmp1;
      i_val = __float_as_int(val + __int_as_float(tmp1));
    }
}

__device__ void cuda_insert_slice(real *model, real *weight, real *slice,
				  int * mask, real w, real *rot, real *x_coordinates,
				  real *y_coordinates, real *z_coordinates, int slice_rows,
				  int slice_cols, int model_x, int model_y, int model_z,
				  int tid, int step)
{
  const int x_max = slice_rows;
  const int y_max = slice_cols;
  //tabulate angle later
  real new_x, new_y, new_z;
  int round_x, round_y, round_z;
  for (int x = 0; x < x_max; x++) {
    for (int y = tid; y < y_max; y+=step) {
      if (mask[y*x_max + x] == 1) {
	/* This is just a matrix multiplication with rot */
	new_x =
	  (rot[0]*rot[0] + rot[1]*rot[1] -
	   rot[2]*rot[2] - rot[3]*rot[3])*x_coordinates[y*x_max+x] +
	  (2.0f*rot[1]*rot[2] -
	   2.0f*rot[0]*rot[3])*y_coordinates[y*x_max+x] +
	  (2.0f*rot[1]*rot[3] +
	   2.0f*rot[0]*rot[2])*z_coordinates[y*x_max+x];
	new_y =
	  (2.0f*rot[1]*rot[2] +
	   2.0f*rot[0]*rot[3])*x_coordinates[y*x_max+x] +
	  (rot[0]*rot[0] - rot[1]*rot[1] +
	   rot[2]*rot[2] - rot[3]*rot[3])*y_coordinates[y*x_max+x] +
	  (2.0f*rot[2]*rot[3] -
	   2.0f*rot[0]*rot[1])*z_coordinates[y*x_max+x];
	new_z =
	  (2.0f*rot[1]*rot[3] -
	   2.0f*rot[0]*rot[2])*x_coordinates[y*x_max+x] +
	  (2.0f*rot[2]*rot[3] +
	   2.0f*rot[0]*rot[1])*y_coordinates[y*x_max+x] +
	  (rot[0]*rot[0] - rot[1]*rot[1] -
	   rot[2]*rot[2] + rot[3]*rot[3])*z_coordinates[y*x_max+x];
	round_x = roundf(model_x/2.0f + 0.5f + new_x);
	round_y = roundf(model_y/2.0f + 0.5f + new_y);
	round_z = roundf(model_z/2.0f + 0.5f + new_z);
	if (round_x >= 0 && round_x < model_x &&
	    round_y >= 0 && round_y < model_y &&
	    round_z >= 0 && round_z < model_z) {
	  /* this is a simple compile time check that can go bad at runtime, but such is life */
#if __CUDA_ARCH__ >= 200
	  atomicAdd(&model[(int)(round_z*model_x*model_y + round_y*model_x + round_x)], w * slice[y*x_max + x]);
	  atomicAdd(&weight[(int)(round_z*model_x*model_y + round_y*model_x + round_x)], w);
#else
	  atomicFloatAdd(&model[(int)(round_z*model_x*model_y + round_y*model_x + round_x)], w * slice[y*x_max + x]);
	  atomicFloatAdd(&weight[(int)(round_z*model_x*model_y + round_y*model_x + round_x)], w);
#endif
	  //	  model[(round_z*model_x*model_y + round_y*model_x + round_x)] += w * slice[y*x_max + x];	    
	  //	  weight[(round_z*model_x*model_y + round_y*model_x + round_x)] += w;
	}
      }
    }
  }
}


__global__ void update_slices_kernel(real * images, real * slices, int * mask, real * respons,
				     real * scaling, int * active_images, int N_images, int slice_start, int N_2d,
				     real * slices_total_respons, real * rot,
				     real * x_coord, real * y_coord, real * z_coord,
				     real * model, real * weight,
				     int slice_rows, int slice_cols,
				     int model_x, int model_y, int model_z, real * weights){
  /* each block takes care of 1 slice */
  int bid = blockIdx.x;
  int tid = threadIdx.x;
  int step = blockDim.x;
  real total_respons = 0.0f;
  int i_slice = bid;
  for (int i = tid; i < N_2d; i+=step) {
    if (mask[i] != 0) {
      real sum = 0;
      for (int i_image = 0; i_image < N_images; i_image++) {
	if (active_images[i_image]) {
	  sum += images[i_image*N_2d+i]*
	    respons[(slice_start+i_slice)*N_images+i_image]/scaling[i_image];
	}
      }
      slices[i_slice*N_2d+i] = sum;
    }
  }
  for (int i_image = 0; i_image < N_images; i_image++) {
    if (active_images[i_image]) {
      total_respons += respons[(slice_start+i_slice)*N_images+i_image];
    }
  }
  if(tid == 0){    
    slices_total_respons[bid] =  total_respons;
  }  
  if(total_respons > 1e-10f){
    for (int i = tid; i < N_2d; i+=step) {
      if (mask[i] != 0) {
	slices[i_slice*N_2d+i] /= total_respons;
      }
    }
  }
  
}

__global__ void insert_slices_kernel(real * images, real * slices, int * mask, real * respons,
				     real * scaling, int N_images, int N_2d,
				     real * slices_total_respons, real * rot,
				     real * x_coord, real * y_coord, real * z_coord,
				     real * model, real * weight,
				     int slice_rows, int slice_cols,
				     int model_x, int model_y, int model_z, real * weights){
  /* each block takes care of 1 slice */
  int bid = blockIdx.x;
  int tid = threadIdx.x;
  int step = blockDim.x;
  int i_slice = bid;
  real total_respons = slices_total_respons[bid];
  if(total_respons > 1e-10f){
    cuda_insert_slice(model,weight,&slices[i_slice*N_2d],mask,weights[i_slice]*total_respons,
		      &rot[4*i_slice],x_coord,y_coord,z_coord,
		      slice_rows,slice_cols,model_x,model_y,model_z,tid,step);
  }  
}

template<typename T>
__device__ void inblock_reduce(T * data){
  __syncthreads();
  for(unsigned int s=blockDim.x/2; s>0; s>>=1){
    if (threadIdx.x < s){
      data[threadIdx.x] += data[threadIdx.x + s];
    }
    __syncthreads();
  }  
}

__device__ void cuda_calculate_responsability_absolute_atomic(float *slice, float *image, int *mask, real sigma, real scaling, int N_2d, int tid, int step, real * sum_cache, int * count_cache)
{
  real sum = 0.0;
  const int i_max = N_2d;
  int count = 0;
  for (int i = tid; i < i_max; i+=step) {
    if (mask[i] != 0 && slice[i] >= 0.0f) {
      sum += pow(slice[i] - image[i]/scaling,2);
      count++;
    }
  }
  sum_cache[tid] = sum;
  count_cache[tid] = count;
  //  return -sum/2.0/(real)count/pow(sigma,2); //return in log scale.
}


__global__ void calculate_fit_kernel(real *slices, real *images, int *mask, real *respons, real *fit, real sigma, real *scaling, int N_2d, int slice_start){
  __shared__ real sum_cache[256];
  __shared__ int count_cache[256];
  int tid = threadIdx.x;
  int step = blockDim.x;
  int i_image = blockIdx.x;
  int i_slice = blockIdx.y;
  int N_images = gridDim.x;
  cuda_calculate_responsability_absolute_atomic(&slices[i_slice*N_2d],
					 &images[i_image*N_2d],mask,
					 sigma,scaling[i_image], N_2d, tid,step,
					 sum_cache,count_cache);
  inblock_reduce(sum_cache);
  inblock_reduce(count_cache);
  
  if(tid == 0){
    atomicFloatAdd(&fit[i_image], expf(-sum_cache[0]/2.0/(real)count_cache[0]/pow(sigma,2)) *
		   respons[(slice_start+i_slice)*N_images+i_image]);
  }
  
}
