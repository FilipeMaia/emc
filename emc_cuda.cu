#include "hip/hip_runtime.h"
#include "emc.h"
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <thrust/iterator/constant_iterator.h>
#include <thrust/fill.h>


__global__ void update_slices_kernel(real * images, real * slices, int * mask, real * respons,
				     real * scaling, int * active_images, int N_images, int slice_start, int N_2d,
				     real * slices_total_respons, real * rot,
				     real * x_coord, real * y_coord, real * z_coord,
				     real * model, real * weight,
				     int slice_rows, int slice_cols,
				     int model_x, int model_y, int model_z, real * weights);

__global__ void insert_slices_kernel(real * images, real * slices, int * mask, real * respons,
				     real * scaling, int N_images, int N_2d,
				     real * slices_total_respons, real * rot,
				     real * x_coord, real * y_coord, real * z_coord,
				     real * model, real * weight,
				     int slice_rows, int slice_cols,
				     int model_x, int model_y, int model_z, real * weights);

__global__ void calculate_fit_kernel(real *slices, real *images, int *mask,
				     real *respons, real *fit, real sigma,
				     real *scaling, int N_2d, int slice_start);

template<typename T>
__device__ void inblock_reduce(T * data){
  __syncthreads();
  for(unsigned int s=blockDim.x/2; s>0; s>>=1){
    if (threadIdx.x < s){
      data[threadIdx.x] += data[threadIdx.x + s];
    }
    __syncthreads();
  }  
}

template<typename T>
__device__ void inblock_reduce_y(T * data){
  __syncthreads();
  for(unsigned int s=blockDim.y/2; s>0; s>>=1){
    if (threadIdx.y < s){
      data[threadIdx.y] += data[threadIdx.y+s];
    }
    __syncthreads();
  }
}

template<typename T>
__device__ void inblock_maximum(T * data){
  __syncthreads();
  for(unsigned int s=blockDim.x/2; s>0; s>>=1){
    if (threadIdx.x < s){
      if(data[threadIdx.x] < data[threadIdx.x + s]){
	data[threadIdx.x] = data[threadIdx.x + s];
      }
    }
    __syncthreads();
  }  
}



__device__ void cuda_get_slice(real *model, real *slice,
			       real *rot, real *x_coordinates,
			       real *y_coordinates, real *z_coordinates, int slice_rows,
			       int slice_cols, int model_x, int model_y, int model_z,
			       int tid, int step)
{
  const int x_max = slice_rows;
  const int y_max = slice_cols;
  //tabulate angle later
  real new_x, new_y, new_z;
  int round_x, round_y, round_z;
  for (int x = 0; x < x_max; x++) {
    for (int y = tid; y < y_max; y+=step) {
      /* This is just a matrix multiplication with rot */
      new_x =
	(rot[0]*rot[0] + rot[1]*rot[1] -
	 rot[2]*rot[2] - rot[3]*rot[3])*x_coordinates[y*x_max+x] +
	(2.0f*rot[1]*rot[2] -
	 2.0f*rot[0]*rot[3])*y_coordinates[y*x_max+x] +
	(2.0f*rot[1]*rot[3] +
	 2.0f*rot[0]*rot[2])*z_coordinates[y*x_max+x];
      new_y =
	(2.0f*rot[1]*rot[2] +
	 2.0f*rot[0]*rot[3])*x_coordinates[y*x_max+x] +
	(rot[0]*rot[0] - rot[1]*rot[1] +
	 rot[2]*rot[2] - rot[3]*rot[3])*y_coordinates[y*x_max+x] +
	(2.0f*rot[2]*rot[3] -
	 2.0f*rot[0]*rot[1])*z_coordinates[y*x_max+x];
      new_z =
	(2.0f*rot[1]*rot[3] -
	 2.0f*rot[0]*rot[2])*x_coordinates[y*x_max+x] +
	(2.0f*rot[2]*rot[3] +
	 2.0f*rot[0]*rot[1])*y_coordinates[y*x_max+x] +
	(rot[0]*rot[0] - rot[1]*rot[1] -
	 rot[2]*rot[2] + rot[3]*rot[3])*z_coordinates[y*x_max+x];
      round_x = roundf(model_x/2.0f + 0.5f + new_x);
      round_y = roundf(model_y/2.0f + 0.5f + new_y);
      round_z = roundf(model_z/2.0f + 0.5f + new_z);
      if (round_x > 0 && round_x < model_x &&
	  round_y > 0 && round_y < model_y &&
	  round_z > 0 && round_z < model_z) {
	slice[y*x_max+x] = model[(round_z*model_x*model_y + round_y*model_x + round_x)];
      }else{
	slice[y*x_max+x] = 0.0f;
      }
    }
  }
}

/* updated to use rotations with an offset start. */
__global__ void get_slices_kernel(real * model, real * slices, real *rot, real *x_coordinates,
				  real *y_coordinates, real *z_coordinates, int slice_rows,
				  int slice_cols, int model_x, int model_y, int model_z,
				  int start_slice){
  int bid = blockIdx.x;
  int i_slice = bid;
  int tid = threadIdx.x;
  int step = blockDim.x;
  int N_2d = slice_rows*slice_cols;
  cuda_get_slice(model,&slices[N_2d*i_slice],&rot[4*(start_slice+i_slice)],x_coordinates,
		 y_coordinates,z_coordinates,slice_rows,slice_cols,model_x,model_y,
		 model_z,tid,step);
}

/* This responsability does not yet take scaling of patterns into accoutnt. */
__device__ void cuda_calculate_responsability_absolute(float *slice, float *image, int *mask, real sigma, real scaling, int N_2d, int tid, int step, real * sum_cache, int * count_cache)
{
  real sum = 0.0;
  const int i_max = N_2d;
  int count = 0;
  for (int i = tid; i < i_max; i+=step) {
    if (mask[i] != 0 && slice[i] >= 0.0f) {
      sum += pow(slice[i] - image[i]/scaling,2);
      count++;
    }
  }
  sum_cache[tid] = sum;
  count_cache[tid] = count;
  //  return -sum/2.0/(real)count/pow(sigma,2); //return in log scale.
}

/* Now takes a starting slice. Otherwise unchanged */
__global__ void calculate_responsabilities_kernel(float * slices, float * images, int * mask,
						  real sigma, real * scaling, real * respons, 
						  int N_2d, int slice_start){
  __shared__ real sum_cache[256];
  __shared__ int count_cache[256];
  int tid = threadIdx.x;
  int step = blockDim.x;
  int i_image = blockIdx.x;
  int i_slice = blockIdx.y;
  int N_images = gridDim.x;
  cuda_calculate_responsability_absolute(&slices[i_slice*N_2d],
					 &images[i_image*N_2d],mask,
					 sigma,scaling[i_image], N_2d, tid,step,
					 sum_cache,count_cache);
  inblock_reduce(sum_cache);
  inblock_reduce(count_cache);
  
  if(tid == 0){
    respons[(slice_start+i_slice)*N_images+i_image] = -sum_cache[0]/2.0/(real)count_cache[0]/pow(sigma,2);
  }   
}


/* Now takes start slice and slice chunk. Also removed memcopy, done separetely later. */
void cuda_calculate_responsabilities(real * d_slices, real * d_images, int * d_mask,
				     real sigma, real * d_scaling, real * d_respons, 
				     int N_2d, int N_images, int slice_start, int slice_chunk){
  hipEvent_t k_begin;
  hipEvent_t k_end;
  hipEventCreate(&k_begin);
  hipEventCreate(&k_end);
  hipEventRecord (k_begin,0);

  dim3 nblocks(N_images,slice_chunk);
  int nthreads = 256;
  calculate_responsabilities_kernel<<<nblocks,nthreads>>>(d_slices,d_images,d_mask,
							  sigma,d_scaling,d_respons,
							  N_2d, slice_start);
  hipError_t status = hipGetLastError();
  if(status != hipSuccess){
    printf("CUDA Error (calc resp): %s\n",hipGetErrorString(status));
  }

  hipEventRecord(k_end,0);
  hipEventSynchronize(k_end);
  real k_ms;
  hipEventElapsedTime (&k_ms, k_begin, k_end);
  //printf("cuda calculate_responsabilities time = %fms\n",k_ms);
}
  
void cuda_calculate_responsabilities_sum(real * respons, real * d_respons, int N_slices,
					 int N_images){
  hipMemcpy(respons,d_respons,sizeof(real)*N_slices*N_images,hipMemcpyDeviceToHost);
  real respons_sum = 0;
  for(int i = 0;i<N_slices*N_images;i++){
    respons_sum += respons[i];
  }
  printf("respons_sum = %f\n",respons_sum);
  hipError_t status = hipGetLastError();
  if(status != hipSuccess){
    printf("CUDA Error (resp sum): %s\n",hipGetErrorString(status));
  }
}  

__global__ void calculate_weighted_power_kernel(real * images, real * slices, int * mask,
						real *respons, real * weighted_power, int N_images,
						int slice_start, int slice_chunk, int N_2d) {
  __shared__ real correlation[256];
  //__shared__ int count[256];
  int step = blockDim.x;
  int bid = blockIdx.x;
  int tid = threadIdx.x;
  int i_image = bid;
  for (int i_slice = 0; i_slice < slice_chunk; i_slice++) { 
    correlation[tid] = 0.0;
    //count[tid] = 0;
    for (int i = tid; i < N_2d; i+=step) {
      if (mask[i] != 0 && slices[i_slice*N_2d+i] > 0.0f) {
	correlation[tid] += images[i_image*N_2d+i]*slices[i_slice*N_2d+i];
	//correlation[tid] += images[i_image*N_2d+i]/slices[i_slice*N_2d+i];
	//count[tid] += 1;
      }
    }
    inblock_reduce(correlation);
    //inblock_reduce(count);
    if(tid == 0){
      weighted_power[i_image] += respons[(slice_start+i_slice)*N_images+i_image]*correlation[tid];
      //weighted_power[i_image] += correlation[tid]/count[tid]*respons[(slice_start+i_slice)*N_images+i_image];
    }
  }
}

__global__ void slice_weighting_kernel(real * images,int * mask,
				       real * scaling, real *weighted_power,
				       int N_slices, int N_2d){
  __shared__ real image_power[256];
  int bid = blockIdx.x;
  int tid = threadIdx.x;
  int step = blockDim.x;
  int i_image = bid;  
  // make sure weighted power is set to 0


  image_power[tid] = 0.0;
  for (int i = tid; i < N_2d; i+=step) {
    if (mask[i] != 0) {
      image_power[tid] += pow(images[i_image*N_2d+i],2);
    }
  }
  inblock_reduce(image_power);

  if(tid == 0){
    scaling[i_image] = image_power[tid]/weighted_power[i_image];
    //scaling[i_image] = weighted_power[i_image];
  }
}

void cuda_update_weighted_power(real * d_images, real * d_slices, int * d_mask,
				real * d_respons, real * d_weighted_power, int N_images,
				int slice_start, int slice_chunk, int N_2d) {
  hipEvent_t k_begin;
  hipEvent_t k_end;
  hipEventCreate(&k_begin);
  hipEventCreate(&k_end);
  hipEventRecord (k_begin,0);

  int nblocks = N_images;
  int nthreads = 256;
  calculate_weighted_power_kernel<<<nblocks,nthreads>>>(d_images,d_slices,d_mask,
							d_respons,d_weighted_power, N_images,
						     slice_start,slice_chunk,N_2d);
  hipError_t status = hipGetLastError();
  if(status != hipSuccess){
    printf("CUDA Error: %s\n",hipGetErrorString(status));
  }

  hipEventRecord(k_end,0);
  hipEventSynchronize(k_end);
  real k_ms;
  hipEventElapsedTime (&k_ms, k_begin, k_end);
  //printf("cuda calculate weighted power time = %fms\n",k_ms);
}

void cuda_update_scaling(real * d_images, int * d_mask,
			 real * d_scaling, real *d_weighted_power, int N_images,
			 int N_slices, int N_2d, real * scaling){
  hipEvent_t begin;
  hipEvent_t end;
  hipEventCreate(&begin);
  hipEventCreate(&end);
  hipEventRecord (begin,0);
  int nblocks = N_images;
  int nthreads = 256;
  hipEvent_t k_begin;
  hipEvent_t k_end;
  hipEventCreate(&k_begin);
  hipEventCreate(&k_end);
  hipEventRecord (k_begin,0);
  slice_weighting_kernel<<<nblocks,nthreads>>>(d_images,d_mask,d_scaling,
					       d_weighted_power,N_slices,N_2d);
  hipMemcpy(scaling,d_scaling,sizeof(real)*N_images,hipMemcpyDeviceToHost);
  hipEventRecord(k_end,0);
  hipEventSynchronize(k_end);
  real k_ms;
  hipEventElapsedTime (&k_ms, k_begin, k_end);
  //printf("cuda kernel update scaling time = %fms\n",k_ms);

  hipError_t status = hipGetLastError();
  if(status != hipSuccess){
    printf("CUDA Error (update scaling): %s\n",hipGetErrorString(status));
  }
  hipEventRecord(end,0);
  hipEventSynchronize (end);
  real ms;
  hipEventElapsedTime (&ms, begin, end);
  //printf("cuda update scaling time = %fms\n",ms);
}

/* function now takes a start slice and a number of slices to retrieve */
void cuda_get_slices(sp_3matrix * model, real * d_model, real * d_slices, real * d_rot, 
		     real * d_x_coordinates, real * d_y_coordinates,
		     real * d_z_coordinates, int start_slice, int slice_chunk){
  hipEvent_t k_begin;
  hipEvent_t k_end;
  hipEventCreate(&k_begin);
  hipEventCreate(&k_end);
  hipEventRecord (k_begin,0);

  int rows = sp_3matrix_x(model);
  int cols = sp_3matrix_y(model);
  int N_2d = sp_3matrix_x(model)*sp_3matrix_y(model);
  int nblocks = slice_chunk;
  int nthreads = 256;
  get_slices_kernel<<<nblocks,nthreads>>>(d_model, d_slices, d_rot,d_x_coordinates,
					  d_y_coordinates,d_z_coordinates,
					  rows,cols,
					  sp_3matrix_x(model),sp_3matrix_y(model),
					  sp_3matrix_z(model), start_slice);
  hipError_t status = hipGetLastError();
  if(status != hipSuccess){
    printf("CUDA Error (get slices): %s\n",hipGetErrorString(status));
  }

  hipEventRecord(k_end,0);
  hipEventSynchronize(k_end);
  real k_ms;
  hipEventElapsedTime (&k_ms, k_begin, k_end);
  //printf("cuda calculate slice time = %fms\n",k_ms);
}

void cuda_update_slices(real * d_images, real * d_slices, int * d_mask,
			real * d_respons, real * d_scaling, int * d_active_images, int N_images,
			int slice_start, int slice_chunk, int N_2d,
			sp_3matrix * model, real * d_model,
			real *d_x_coordinates, real *d_y_coordinates,
			real *d_z_coordinates, real *d_rot, real * weights,
			real * d_weight, sp_matrix ** images){
  dim3 nblocks = slice_chunk;//N_slices;
  int nthreads = 256;
  real * d_slices_total_respons;
  hipMalloc(&d_slices_total_respons,sizeof(real)*slice_chunk);

  real * d_weights;
  hipMalloc(&d_weights,sizeof(real)*slice_chunk);
  hipMemcpy(d_weights,weights,sizeof(real)*slice_chunk,hipMemcpyHostToDevice);

  hipEvent_t k_begin;
  hipEvent_t k_end;
  hipEventCreate(&k_begin);
  hipEventCreate(&k_end);
  hipEventRecord (k_begin,0);

  update_slices_kernel<<<nblocks,nthreads>>>(d_images, d_slices, d_mask, d_respons,
					     d_scaling, d_active_images, N_images, slice_start, N_2d,
					     d_slices_total_respons, d_rot,d_x_coordinates,
					     d_y_coordinates,d_z_coordinates,d_model, d_weight,
					     sp_matrix_rows(images[0]),sp_matrix_cols(images[0]),
					     sp_3matrix_x(model),sp_3matrix_y(model),
					     sp_3matrix_z(model),d_weights);  
  hipDeviceSynchronize();
  insert_slices_kernel<<<nblocks,nthreads>>>(d_images, d_slices, d_mask, d_respons,
					     d_scaling, N_images, N_2d,
					     d_slices_total_respons, d_rot,d_x_coordinates,
					     d_y_coordinates,d_z_coordinates,d_model, d_weight,
					     sp_matrix_rows(images[0]),sp_matrix_cols(images[0]),
					     sp_3matrix_x(model),sp_3matrix_y(model),
					     sp_3matrix_z(model),d_weights);  
  hipEventRecord(k_end,0);
  hipEventSynchronize(k_end);
  real k_ms;
  hipEventElapsedTime (&k_ms, k_begin, k_end);
  //printf("cuda kernel slice update time = %fms\n",k_ms);

  hipError_t status = hipGetLastError();
  if(status != hipSuccess){
    printf("CUDA Error (update slices): %s\n",hipGetErrorString(status));
  }
}

real cuda_model_max(real * model, int model_size){
  thrust::device_ptr<real> p(model);
  real max = thrust::reduce(p, p+model_size, real(0), thrust::maximum<real>());
  return max;
}

void cuda_allocate_slices(real ** slices, int side, int N_slices){
  hipMalloc(slices,sizeof(real)*side*side*N_slices);  
}

void cuda_allocate_model(real ** d_model, sp_3matrix * model){
  hipMalloc(d_model,sizeof(real)*sp_3matrix_size(model));
  hipMemcpy(*d_model,model->data,sizeof(real)*sp_3matrix_size(model),hipMemcpyHostToDevice);
}

void cuda_allocate_mask(int ** d_mask, sp_imatrix * mask){
  hipMalloc(d_mask,sizeof(int)*sp_imatrix_size(mask));
  hipMemcpy(*d_mask,mask->data,sizeof(int)*sp_imatrix_size(mask),hipMemcpyHostToDevice);
}

void cuda_allocate_rotations(real ** d_rotations, Quaternion ** rotations,  int N_slices){

  hipMalloc(d_rotations,sizeof(real)*4*N_slices);
  for(int i = 0;i<N_slices;i++){
    hipMemcpy(&(*d_rotations)[4*i],rotations[i]->q,sizeof(real)*4,hipMemcpyHostToDevice);
  }
}

void cuda_allocate_images(real ** d_images, sp_matrix ** images,  int N_images){

  hipMalloc(d_images,sizeof(real)*sp_matrix_size(images[0])*N_images);
  for(int i = 0;i<N_images;i++){
    hipMemcpy(&(*d_images)[sp_matrix_size(images[0])*i],images[i]->data,sizeof(real)*sp_matrix_size(images[0]),hipMemcpyHostToDevice);
  }
}

void cuda_allocate_coords(real ** d_x, real ** d_y, real ** d_z, sp_matrix * x,
			  sp_matrix * y, sp_matrix * z){
  hipMalloc(d_x,sizeof(real)*sp_matrix_size(x));
  hipMalloc(d_y,sizeof(real)*sp_matrix_size(x));
  hipMalloc(d_z,sizeof(real)*sp_matrix_size(x));
  hipMemcpy(*d_x,x->data,sizeof(real)*sp_matrix_size(x),hipMemcpyHostToDevice);
  hipMemcpy(*d_y,y->data,sizeof(real)*sp_matrix_size(x),hipMemcpyHostToDevice);
  hipMemcpy(*d_z,z->data,sizeof(real)*sp_matrix_size(x),hipMemcpyHostToDevice);
}

void cuda_reset_model(sp_3matrix * model, real * d_model){
  hipMemset(d_model,0,sizeof(real)*sp_3matrix_size(model));
}

void cuda_copy_model(sp_3matrix * model, real *d_model){
  hipMemcpy(model->data,d_model,sizeof(real)*sp_3matrix_size(model),hipMemcpyDeviceToHost);
}

__global__ void cuda_normalize_model_kernel(real * model, real * weight, int n){
  int i = threadIdx.x + blockIdx.x*blockDim.x;
  if(weight[i] > 0.0f){
    model[i] /= weight[i];
  }else{
    model[i] = 0.0f;
  }
}

__global__ void cuda_mask_out_model_kernel(real *model, real *weight, int n){
  int i = threadIdx.x + blockIdx.x*blockDim.x;
  if(weight[i] <= 0.0f){
    model[i] = -1.0f;
  }
}

void cuda_normalize_model(sp_3matrix * model, real * d_model, real * d_weight){
  int n = sp_3matrix_size(model);
  int nthreads = 256;
  int nblocks = (n+nthreads-1)/nthreads;
  cuda_normalize_model_kernel<<<nblocks,nthreads>>>(d_model,d_weight,n);
  hipDeviceSynchronize();
  thrust::device_ptr<real> p(d_model);
  real model_sum = thrust::reduce(p, p+n, real(0), thrust::plus<real>());
  model_sum /= n;
  /* model /= model_sum; */
  thrust::transform(p, p+n,thrust::make_constant_iterator(1.0f/model_sum), p, thrust::multiplies<real>()); 
  cuda_mask_out_model_kernel<<<nblocks,nthreads>>>(d_model,d_weight,n);
}

void cuda_allocate_real(real ** x, int n){
  hipMalloc(x,n);
}

void cuda_allocate_int(int ** x, int n){
  hipMalloc(x,n);
}

void cuda_set_to_zero(real * x, int n){
  hipMemset(x,0.0,sizeof(real)*n);
}

void cuda_copy_real_to_device(real *x, real *d_x, int n){
  hipMemcpy(d_x,x,n*sizeof(real),hipMemcpyHostToDevice);
}

void cuda_copy_real_to_host(real *x, real *d_x, int n){
  hipMemcpy(x,d_x,n*sizeof(real),hipMemcpyDeviceToHost);
}

void cuda_copy_int_to_device(int *x, int *d_x, int n){
  hipMemcpy(d_x,x,n*sizeof(int),hipMemcpyHostToDevice);
}

void cuda_copy_int_to_host(int *x, int *d_x, int n){
  hipMemcpy(x,d_x,n*sizeof(int),hipMemcpyDeviceToHost);
}
			  
void cuda_allocate_scaling(real ** d_scaling, int N_images){
  hipMalloc(d_scaling,N_images*sizeof(real));
  thrust::device_ptr<real> p(*d_scaling);
  thrust::fill(p, p+N_images, real(1));
}

__global__ void cuda_normalize_responsabilities_kernel(real * respons, int N_slices, int N_images){
  __shared__ real cache[256];
  int i_image = blockIdx.x;
  int tid = threadIdx.x;
  int step = blockDim.x;
  cache[tid] = -1.0e10f;
  for(int i_slice = tid;i_slice < N_slices;i_slice += step){
    if(cache[tid] < respons[i_slice*N_images+i_image]){
      cache[tid] = respons[i_slice*N_images+i_image];
    }
  }
  inblock_maximum(cache);
  real max_resp = cache[0];
  for (int i_slice = tid; i_slice < N_slices; i_slice+= step) {
    respons[i_slice*N_images+i_image] -= max_resp;
  }  
  cache[tid] = 0;
  for (int i_slice = tid; i_slice < N_slices; i_slice+=step) {
    if (respons[i_slice*N_images+i_image] > -1.0e10f) {
      respons[i_slice*N_images+i_image] = expf(respons[i_slice*N_images+i_image]);
      cache[tid] += respons[i_slice*N_images+i_image];
    } else {
      respons[i_slice*N_images+i_image] = 0.0f;
    }
  }
  inblock_reduce(cache);
  real sum = cache[0];
  for (int i_slice = tid; i_slice < N_slices; i_slice+=step) {
    respons[i_slice*N_images+i_image] /= sum;
  }
}

void cuda_normalize_responsabilities(real * d_respons, int N_slices, int N_images){
  int nblocks = N_images;
  int nthreads = 256;
  cuda_normalize_responsabilities_kernel<<<nblocks,nthreads>>>(d_respons, N_slices, N_images);
  hipError_t status = hipGetLastError();
  if(status != hipSuccess){
    printf("CUDA Error (norm resp): %s\n",hipGetErrorString(status));
  }
}

// x_log_x<T> computes the f(x) -> x*log(x)
template <typename T>
struct x_log_x
{
  __host__ __device__
  T operator()(const T& x) const { 
    if(x > 0){
      return x * logf(x);
    }else{
      return 0;
    }
  }
};

real cuda_total_respons(real * d_respons, real * respons,int n){
  thrust::device_ptr<real> p(d_respons);
  x_log_x<real> unary_op;
  thrust::plus<real> binary_op;
  real init = 0;
  // Calculates sum_0^n d_respons*log(d_respons)
  return thrust::transform_reduce(p, p+n, unary_op, init, binary_op);
}

void cuda_copy_slice_chunk_to_host(real * slices, real * d_slices, int slice_start, int slice_chunk, int N_2d){
  hipEvent_t k_begin;
  hipEvent_t k_end;
  hipEventCreate(&k_begin);
  hipEventCreate(&k_end);
  hipEventRecord (k_begin,0);

  hipMemcpy(&slices[slice_start],d_slices,sizeof(real)*N_2d*slice_chunk,hipMemcpyDeviceToHost);

  hipEventRecord(k_end,0);
  hipEventSynchronize(k_end);
  real k_ms;
  hipEventElapsedTime (&k_ms, k_begin, k_end);
  //printf("cuda copy slice to host time = %fms\n",k_ms);

}

void cuda_copy_slice_chunk_to_device(real * slices, real * d_slices, int slice_start, int slice_chunk, int N_2d){
  hipEvent_t k_begin;
  hipEvent_t k_end;
  hipEventCreate(&k_begin);
  hipEventCreate(&k_end);
  hipEventRecord (k_begin,0);

  hipMemcpy(d_slices,&slices[slice_start],sizeof(real)*N_2d*slice_chunk,hipMemcpyHostToDevice);

  hipEventRecord(k_end,0);
  hipEventSynchronize(k_end);
  real k_ms;
  hipEventElapsedTime (&k_ms, k_begin, k_end);
  //printf("cuda copy slice to device time = %fms\n",k_ms);

}

void cuda_calculate_fit(real * slices, real * d_images, int * d_mask,
			real * d_scaling, real * d_respons, real * d_fit, real sigma,
			int N_2d, int N_images, int slice_start, int slice_chunk){
  //call the kernel  
  dim3 nblocks(N_images,slice_chunk);
  int nthreads = 256;
  calculate_fit_kernel<<<nblocks,nthreads>>>(slices, d_images, d_mask,
					     d_respons, d_fit, sigma, d_scaling,
					     N_2d, slice_start);

}

