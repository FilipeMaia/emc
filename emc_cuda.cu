#include "hip/hip_runtime.h"
#include "emc.h"

template<typename T>
__device__ void inblock_reduce(T * data){
  __syncthreads();
  for(unsigned int s=blockDim.x/2; s>0; s>>=1){
    if (threadIdx.x < s){
      data[threadIdx.x] += data[threadIdx.x + s];
    }
    __syncthreads();
  }  
}

/* This responsability does not yet take scaling of patterns into accoutnt. */
__device__ void cuda_calculate_responsability_absolute(float *slice, float *image, int *mask, real sigma, real scaling, int N_2d, int tid, int step, real * sum_cache, int * count_cache)
{
  real sum = 0.0;
  const int i_max = N_2d;
  int count = 0;
  for (int i = tid; i < i_max; i+=step) {
    if (mask[i] != 0) {
      sum += pow(slice[i] - image[i]/scaling,2);
      count++;
    }
  }
  sum_cache[tid] = sum;
  count_cache[tid] = count;
  //  return -sum/2.0/(real)count/pow(sigma,2); //return in log scale.
}

__global__ void calculate_responsabilities_kernel(float * slices, float * images, int * mask,
						  real sigma, real * scaling, real * respons, 
						  int N_2d){
  __shared__ real sum_cache[256];
  __shared__ int count_cache[256];
  int tid = threadIdx.x;
  int step = blockDim.x;
  int i_image = blockIdx.x;
  int i_slice = blockIdx.y;
  int N_images = gridDim.x;
  cuda_calculate_responsability_absolute(&slices[i_slice*N_2d],
					 &images[i_image*N_2d],mask,
					 sigma,scaling[i_image], N_2d, tid,step,
					 sum_cache,count_cache);
  inblock_reduce(sum_cache);
  inblock_reduce(count_cache);
  
  if(tid == 0){
    respons[i_slice*N_images+i_image] = -sum_cache[0]/2.0/(real)count_cache[0]/pow(sigma,2);
  }   
}


void cuda_calculate_responsabilities(sp_matrix ** slices, sp_matrix ** images, sp_imatrix * mask,
				     real sigma, real * scaling, real * respons, 
				     int N_2d, int N_images, int N_slices){
  hipEvent_t begin;
  hipEvent_t end;
  hipEventCreate(&begin);
  hipEventCreate(&end);
  hipEventRecord (begin,0);
  real * d_images;
  hipMalloc(&d_images,sizeof(real)*N_2d*N_images);
  for(int i = 0;i<N_images;i++){
    hipMemcpy(&(d_images[i*N_2d]),images[i]->data,sizeof(real)*N_2d,hipMemcpyHostToDevice);
  }
  real * d_slices;
  hipMalloc(&d_slices,sizeof(real)*N_2d*N_slices);
  for(int i = 0;i<N_slices;i++){
    hipMemcpy(&(d_slices[i*N_2d]),slices[i]->data,sizeof(real)*N_2d,hipMemcpyHostToDevice);
  }
  int * d_mask;
  hipMalloc(&d_mask,sizeof(int)*N_2d);
  hipMemcpy(d_mask,mask->data,sizeof(int)*N_2d,hipMemcpyHostToDevice);
  real * d_respons;
  hipMalloc(&d_respons,sizeof(real)*N_slices*N_images);
  hipMemcpy(d_respons,respons,sizeof(real)*N_slices*N_images,hipMemcpyHostToDevice);
  real * d_scaling;
  hipMalloc(&d_scaling,sizeof(real)*N_images);
  hipMemcpy(d_scaling,scaling,sizeof(real)*N_images,hipMemcpyHostToDevice);
  dim3 nblocks(N_images,N_slices);
  int nthreads = 256;
  hipEvent_t k_begin;
  hipEvent_t k_end;
  hipEventCreate(&k_begin);
  hipEventCreate(&k_end);
  hipEventRecord (k_begin,0);
  calculate_responsabilities_kernel<<<nblocks,nthreads>>>(d_slices,d_images,d_mask,
							  sigma,d_scaling,d_respons,
							  N_2d);
  hipEventRecord(k_end,0);
  hipEventSynchronize(k_end);
  real k_ms;
  hipEventElapsedTime (&k_ms, k_begin, k_end);
  printf("cuda kernel calc respons time = %fms\n",k_ms);

  hipError_t status = hipGetLastError();
  if(status != hipSuccess){
    printf("CUDA Error: %s\n",hipGetErrorString(status));
  }
  hipMemcpy(respons,d_respons,sizeof(real)*N_images*N_slices,
	     hipMemcpyDeviceToHost);
  hipFree(d_images);
  hipFree(d_slices);
  hipFree(d_mask);
  hipFree(d_respons);
  hipFree(d_scaling);
  hipEventRecord(end,0);
  hipEventSynchronize (end);
  real ms;
  hipEventElapsedTime (&ms, begin, end);
  printf("cuda calc respons time = %fms\n",ms);
}  

__global__ void slice_weighting_kernel(real * images, real * slices,int * mask,
		     real * respons, real * scaling,
				       int N_slices, int N_2d, int N_images){
  __shared__ real image_power[256];
  __shared__ real correlation[256];
  int bid = blockIdx.x;
  int tid = threadIdx.x;
  int step = blockDim.x;
  int i_image = bid;  
  real weighted_power = 0;

  image_power[tid] = 0.0;
  for (int i = tid; i < N_2d; i+=step) {
    if (mask[i] != 0) {
      image_power[tid] += pow(images[i_image*N_2d+i],2);
    }
  }
  inblock_reduce(image_power);
  for (int i_slice = 0; i_slice < N_slices; i_slice++) { 
    correlation[tid] = 0.0;
    for (int i = tid; i < N_2d; i+=step) {
      if (mask[i] != 0) {
	correlation[tid] += images[i_image*N_2d+i]*slices[i_slice*N_2d+i];
      }
    }
    inblock_reduce(correlation);
    if(tid == 0){
      weighted_power += respons[i_slice*N_images+i_image]*correlation[tid];
    }
  }  
  if(tid == 0){
    scaling[i_image] = image_power[tid]/weighted_power;
  }
}

void cuda_update_scaling(sp_matrix ** images, sp_matrix ** slices, sp_imatrix * mask,
			 real * respons, real * scaling, int N_images, int N_slices, int N_2d){
  hipEvent_t begin;
  hipEvent_t end;
  hipEventCreate(&begin);
  hipEventCreate(&end);
  hipEventRecord (begin,0);
  real * d_images;
  hipMalloc(&d_images,sizeof(real)*N_2d*N_images);
  for(int i = 0;i<N_images;i++){
    hipMemcpy(&(d_images[i*N_2d]),images[i]->data,sizeof(real)*N_2d,hipMemcpyHostToDevice);
  }
  real * d_slices;
  hipMalloc(&d_slices,sizeof(real)*N_2d*N_slices);
  for(int i = 0;i<N_slices;i++){
    hipMemcpy(&(d_slices[i*N_2d]),slices[i]->data,sizeof(real)*N_2d,hipMemcpyHostToDevice);
  }
  int * d_mask;
  hipMalloc(&d_mask,sizeof(int)*N_2d);
  hipMemcpy(d_mask,mask->data,sizeof(int)*N_2d,hipMemcpyHostToDevice);
  real * d_respons;
  hipMalloc(&d_respons,sizeof(real)*N_slices*N_images);
  hipMemcpy(d_respons,respons,sizeof(real)*N_slices*N_images,hipMemcpyHostToDevice);
  real * d_scaling;
  hipMalloc(&d_scaling,sizeof(real)*N_images);
  hipMemcpy(d_scaling,scaling,sizeof(real)*N_images,hipMemcpyHostToDevice);
  int nblocks = N_images;
  int nthreads = 256;
  hipEvent_t k_begin;
  hipEvent_t k_end;
  hipEventCreate(&k_begin);
  hipEventCreate(&k_end);
  hipEventRecord (k_begin,0);
  slice_weighting_kernel<<<nblocks,nthreads>>>(d_images,d_slices,d_mask,
			 d_respons, d_scaling,
			 N_slices,N_2d, N_images);
  hipEventRecord(k_end,0);
  hipEventSynchronize(k_end);
  real k_ms;
  hipEventElapsedTime (&k_ms, k_begin, k_end);
  printf("cuda kernel update scaling time = %fms\n",k_ms);

  hipError_t status = hipGetLastError();
  if(status != hipSuccess){
    printf("CUDA Error: %s\n",hipGetErrorString(status));
  }
  hipMemcpy(scaling,d_scaling,sizeof(real)*N_images,
	     hipMemcpyDeviceToHost);
  hipFree(d_images);
  hipFree(d_slices);
  hipFree(d_mask);
  hipFree(d_respons);
  hipFree(d_scaling);
  hipEventRecord(end,0);
  hipEventSynchronize (end);
  real ms;
  hipEventElapsedTime (&ms, begin, end);
  printf("cuda update scaling time = %fms\n",ms);
}

__global__ void update_slices_kernel(real * images, real * slices, int * mask, real * respons,
			  real * scaling, int N_images, int N_slices, int N_2d,
			  real * slices_total_respons){
  /* each block takes care of 1 slice */
  int bid = blockIdx.x;
  int tid = threadIdx.x;
  int step = blockDim.x;
  real total_respons = 0.0f;
  int i_slice = bid;
  for (int i = tid; i < N_2d; i+=step) {
    if (mask[i] != 0) {
      real sum = 0;
      for (int i_image = 0; i_image < N_images; i_image++) {
	sum += images[i_image*N_2d+i]*
	  respons[i_slice*N_images+i_image]/scaling[i_image];
      }
      slices[i_slice*N_2d+i] = sum;
    }
  }
  for (int i_image = 0; i_image < N_images; i_image++) {
    total_respons += respons[i_slice*N_images+i_image];
  }
  if(tid == 0){    
    slices_total_respons[bid] =  total_respons;
  }  
  if(total_respons > 1e-10f){
    for (int i = tid; i < N_2d; i+=step) {
      if (mask[i] != 0) {
	slices[i_slice*N_2d+i] /= total_respons;
      }
    }
  }
}

real cuda_update_slices(sp_matrix ** images, sp_matrix ** slices, sp_imatrix * mask,
			real * respons, real * scaling, int N_images, int N_slices, int N_2d,
			sp_3matrix * model, sp_matrix *x_coordinates, sp_matrix *y_coordinates,
			sp_matrix *z_coordinates, Quaternion **rotations, real * weights,
			sp_3matrix * weight){
  hipEvent_t begin;
  hipEvent_t end;
  hipEventCreate(&begin);
  hipEventCreate(&end);
  hipEventRecord (begin,0);
  real * d_images;
  hipMalloc(&d_images,sizeof(real)*N_2d*N_images);
  for(int i = 0;i<N_images;i++){
    hipMemcpy(&(d_images[i*N_2d]),images[i]->data,sizeof(real)*N_2d,hipMemcpyHostToDevice);
  }
  real * d_slices;
  hipMalloc(&d_slices,sizeof(real)*N_2d*N_slices);
  hipMemset(d_slices,0,sizeof(real)*N_2d*N_slices);
  int * d_mask;
  hipMalloc(&d_mask,sizeof(int)*N_2d);
  hipMemcpy(d_mask,mask->data,sizeof(int)*N_2d,hipMemcpyHostToDevice);
  real * d_respons;
  hipMalloc(&d_respons,sizeof(real)*N_slices*N_images);
  hipMemcpy(d_respons,respons,sizeof(real)*N_slices*N_images,hipMemcpyHostToDevice);
  real * d_scaling;
  hipMalloc(&d_scaling,sizeof(real)*N_images);
  hipMemcpy(d_scaling,scaling,sizeof(real)*N_images,hipMemcpyHostToDevice);
  int nblocks = N_slices;
  int nthreads = 256;
  real * d_slices_total_respons;
  hipMalloc(&d_slices_total_respons,sizeof(real)*N_slices);
  hipEvent_t k_begin;
  hipEvent_t k_end;
  hipEventCreate(&k_begin);
  hipEventCreate(&k_end);
  hipEventRecord (k_begin,0);

  update_slices_kernel<<<nblocks,nthreads>>>(d_images, d_slices, d_mask, d_respons,
					     d_scaling, N_images, N_slices, N_2d,
					     d_slices_total_respons);
  hipEventRecord(k_end,0);
  hipEventSynchronize(k_end);
  real k_ms;
  hipEventElapsedTime (&k_ms, k_begin, k_end);
  printf("cuda kernel slice update time = %fms\n",k_ms);

  hipError_t status = hipGetLastError();
  if(status != hipSuccess){
    printf("CUDA Error: %s\n",hipGetErrorString(status));
  }
  real slices_total_respons[N_slices];
  hipMemcpy(slices_total_respons,d_slices_total_respons,sizeof(real)*N_slices,
	     hipMemcpyDeviceToHost);
  real overal_respons = 0.0;
  for (int i_slice = 0; i_slice < N_slices; i_slice++) {
    overal_respons += slices_total_respons[i_slice];
    if(slices_total_respons[i_slice] > 1e-10){
      hipMemcpy(slices[i_slice]->data,&(d_slices[i_slice*N_2d]),sizeof(real)*N_2d,
		 hipMemcpyDeviceToHost);
      hipError_t status = hipGetLastError();
      if(status != hipSuccess){
	printf("CUDA Error: %s\n",hipGetErrorString(status));
      }
      insert_slice(model, weight, slices[i_slice], mask, 
		   weights[i_slice]*slices_total_respons[i_slice],
		   rotations[i_slice], x_coordinates, y_coordinates, z_coordinates);
    }
  }
  hipFree(d_images);
  hipFree(d_slices);
  hipFree(d_mask);
  hipFree(d_respons);
  hipFree(d_scaling);
  hipFree(d_slices_total_respons);
  hipEventRecord(end,0);
  hipEventSynchronize (end);
  real ms;
  hipEventElapsedTime (&ms, begin, end);
  printf("cuda slice update time = %fms\n",ms);
  return overal_respons;
}
