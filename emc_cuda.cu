#include "hip/hip_runtime.h"
#include "emc.h"
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <thrust/iterator/constant_iterator.h>
#include <thrust/fill.h>


__global__ void update_slices_kernel(real * images, real * slices, int * mask, real * respons,
				     real * scaling, int N_images, int N_slices, int N_2d,
				     real * slices_total_respons, real * rot,
				     real * x_coord, real * y_coord, real * z_coord,
				     real * model, real * weight,
				     int slice_rows, int slice_cols,
				     int model_x, int model_y, int model_z, real * weights);

__global__ void insert_slices_kernel(real * images, real * slices, int * mask, real * respons,
				     real * scaling, int N_images, int N_slices, int N_2d,
				     real * slices_total_respons, real * rot,
				     real * x_coord, real * y_coord, real * z_coord,
				     real * model, real * weight,
				     int slice_rows, int slice_cols,
				     int model_x, int model_y, int model_z, real * weights);


template<typename T>
__device__ void inblock_reduce(T * data){
  __syncthreads();
  for(unsigned int s=blockDim.x/2; s>0; s>>=1){
    if (threadIdx.x < s){
      data[threadIdx.x] += data[threadIdx.x + s];
    }
    __syncthreads();
  }  
}

template<typename T>
__device__ void inblock_maximum(T * data){
  __syncthreads();
  for(unsigned int s=blockDim.x/2; s>0; s>>=1){
    if (threadIdx.x < s){
      if(data[threadIdx.x] < data[threadIdx.x + s]){
	data[threadIdx.x] = data[threadIdx.x + s];
      }
    }
    __syncthreads();
  }  
}



__device__ void cuda_get_slice(real *model, real *slice,
				  real *rot, real *x_coordinates,
				  real *y_coordinates, real *z_coordinates, int slice_rows,
				  int slice_cols, int model_x, int model_y, int model_z,
				  int tid, int step)
{
  const int x_max = slice_rows;
  const int y_max = slice_cols;
  //tabulate angle later
  real new_x, new_y, new_z;
  int round_x, round_y, round_z;
  for (int x = 0; x < x_max; x++) {
    for (int y = tid; y < y_max; y+=step) {
      /* This is just a matrix multiplication with rot */
      new_x =
	(rot[0]*rot[0] + rot[1]*rot[1] -
	 rot[2]*rot[2] - rot[3]*rot[3])*x_coordinates[y*x_max+x] +
	(2.0f*rot[1]*rot[2] -
	 2.0f*rot[0]*rot[3])*y_coordinates[y*x_max+x] +
	(2.0f*rot[1]*rot[3] +
	 2.0f*rot[0]*rot[2])*z_coordinates[y*x_max+x];
      new_y =
	(2.0f*rot[1]*rot[2] +
	 2.0f*rot[0]*rot[3])*x_coordinates[y*x_max+x] +
	(rot[0]*rot[0] - rot[1]*rot[1] +
	 rot[2]*rot[2] - rot[3]*rot[3])*y_coordinates[y*x_max+x] +
	(2.0f*rot[2]*rot[3] -
	 2.0f*rot[0]*rot[1])*z_coordinates[y*x_max+x];
      new_z =
	(2.0f*rot[1]*rot[3] -
	 2.0f*rot[0]*rot[2])*x_coordinates[y*x_max+x] +
	(2.0f*rot[2]*rot[3] +
	 2.0f*rot[0]*rot[1])*y_coordinates[y*x_max+x] +
	(rot[0]*rot[0] - rot[1]*rot[1] -
	 rot[2]*rot[2] + rot[3]*rot[3])*z_coordinates[y*x_max+x];
      round_x = roundf(model_x/2.0f + 0.5f + new_x);
      round_y = roundf(model_y/2.0f + 0.5f + new_y);
      round_z = roundf(model_z/2.0f + 0.5f + new_z);
      if (round_x > 0 && round_x < model_x &&
	  round_y > 0 && round_y < model_y &&
	  round_z > 0 && round_z < model_z) {
	slice[y*x_max+x] = model[(round_z*model_x*model_y + round_y*model_x + round_x)];
      }else{
	slice[y*x_max+x] = 0.0f;
      }
    }
  }
}

__global__ void get_slices_kernel(real * model, real * slices, real *rot, real *x_coordinates,
				  real *y_coordinates, real *z_coordinates, int slice_rows,
				  int slice_cols, int model_x, int model_y, int model_z){
  int bid = blockIdx.x;
  int i_slice = bid;
  int tid = threadIdx.x;
  int step = blockDim.x;
  int N_2d = slice_rows*slice_cols;
  cuda_get_slice(model,&slices[N_2d*i_slice],&rot[4*i_slice],x_coordinates,
		 y_coordinates,z_coordinates,slice_rows,slice_cols,model_x,model_y,
		 model_z,tid,step);
}

/* This responsability does not yet take scaling of patterns into accoutnt. */
__device__ void cuda_calculate_responsability_absolute(float *slice, float *image, int *mask, real sigma, real scaling, int N_2d, int tid, int step, real * sum_cache, int * count_cache)
{
  real sum = 0.0;
  const int i_max = N_2d;
  int count = 0;
  for (int i = tid; i < i_max; i+=step) {
    if (mask[i] != 0) {
      sum += pow(slice[i] - image[i]/scaling,2);
      count++;
    }
  }
  sum_cache[tid] = sum;
  count_cache[tid] = count;
  //  return -sum/2.0/(real)count/pow(sigma,2); //return in log scale.
}

__global__ void calculate_responsabilities_kernel(float * slices, float * images, int * mask,
						  real sigma, real * scaling, real * respons, 
						  int N_2d){
  __shared__ real sum_cache[256];
  __shared__ int count_cache[256];
  int tid = threadIdx.x;
  int step = blockDim.x;
  int i_image = blockIdx.x;
  int i_slice = blockIdx.y;
  int N_images = gridDim.x;
  cuda_calculate_responsability_absolute(&slices[i_slice*N_2d],
					 &images[i_image*N_2d],mask,
					 sigma,scaling[i_image], N_2d, tid,step,
					 sum_cache,count_cache);
  inblock_reduce(sum_cache);
  inblock_reduce(count_cache);
  
  if(tid == 0){
    respons[i_slice*N_images+i_image] = -sum_cache[0]/2.0/(real)count_cache[0]/pow(sigma,2);
  }   
}



void cuda_calculate_responsabilities(real * d_slices, real * d_images, int * d_mask,
				     real sigma, real * d_scaling, real * d_respons, 
				     int N_2d, int N_images, int N_slices, real * respons){
  hipEvent_t begin;
  hipEvent_t end;
  hipEventCreate(&begin);
  hipEventCreate(&end);
  hipEventRecord (begin,0);
  dim3 nblocks(N_images,N_slices);
  int nthreads = 256;
  hipEvent_t k_begin;
  hipEvent_t k_end;
  hipEventCreate(&k_begin);
  hipEventCreate(&k_end);
  hipEventRecord (k_begin,0);
  calculate_responsabilities_kernel<<<nblocks,nthreads>>>(d_slices,d_images,d_mask,
							  sigma,d_scaling,d_respons,
							  N_2d);
  hipEventRecord(k_end,0);
  hipEventSynchronize(k_end);
  real k_ms;
  hipEventElapsedTime (&k_ms, k_begin, k_end);
  printf("cuda kernel calc respons time = %fms\n",k_ms);

  hipError_t status = hipGetLastError();
  if(status != hipSuccess){
    printf("CUDA Error: %s\n",hipGetErrorString(status));
  }
  hipMemcpy(respons,d_respons,sizeof(real)*N_slices*N_images,hipMemcpyDeviceToHost);
  real respons_sum = 0;
  for(int i = 0;i<N_slices*N_images;i++){
    respons_sum += respons[i];
  }
  printf("respons_sum = %f\n",respons_sum);
  hipEventRecord(end,0);
  hipEventSynchronize (end);
  real ms;
  hipEventElapsedTime (&ms, begin, end);
  printf("cuda calc respons time = %fms\n",ms);
}  

__global__ void slice_weighting_kernel(real * images, real * slices,int * mask,
		     real * respons, real * scaling,
				       int N_slices, int N_2d, int N_images){
  __shared__ real image_power[256];
  __shared__ real correlation[256];
  int bid = blockIdx.x;
  int tid = threadIdx.x;
  int step = blockDim.x;
  int i_image = bid;  
  real weighted_power = 0;

  image_power[tid] = 0.0;
  for (int i = tid; i < N_2d; i+=step) {
    if (mask[i] != 0) {
      image_power[tid] += pow(images[i_image*N_2d+i],2);
    }
  }
  inblock_reduce(image_power);
  for (int i_slice = 0; i_slice < N_slices; i_slice++) { 
    correlation[tid] = 0.0;
    for (int i = tid; i < N_2d; i+=step) {
      if (mask[i] != 0) {
	correlation[tid] += images[i_image*N_2d+i]*slices[i_slice*N_2d+i];
      }
    }
    inblock_reduce(correlation);
    if(tid == 0){
      weighted_power += respons[i_slice*N_images+i_image]*correlation[tid];
    }
  }  
  if(tid == 0){
    scaling[i_image] = image_power[tid]/weighted_power;
  }
}

void cuda_update_scaling(real * d_images, real * d_slices, int * d_mask,
			 real * d_respons, real * d_scaling, int N_images, int N_slices, int N_2d,
			 real * scaling){
  hipEvent_t begin;
  hipEvent_t end;
  hipEventCreate(&begin);
  hipEventCreate(&end);
  hipEventRecord (begin,0);
  int nblocks = N_images;
  int nthreads = 256;
  hipEvent_t k_begin;
  hipEvent_t k_end;
  hipEventCreate(&k_begin);
  hipEventCreate(&k_end);
  hipEventRecord (k_begin,0);
  slice_weighting_kernel<<<nblocks,nthreads>>>(d_images,d_slices,d_mask,
			 d_respons, d_scaling,
			 N_slices,N_2d, N_images);
  hipMemcpy(scaling,d_scaling,sizeof(real)*N_images,hipMemcpyDeviceToHost);
  hipEventRecord(k_end,0);
  hipEventSynchronize(k_end);
  real k_ms;
  hipEventElapsedTime (&k_ms, k_begin, k_end);
  printf("cuda kernel update scaling time = %fms\n",k_ms);

  hipError_t status = hipGetLastError();
  if(status != hipSuccess){
    printf("CUDA Error: %s\n",hipGetErrorString(status));
  }
  hipEventRecord(end,0);
  hipEventSynchronize (end);
  real ms;
  hipEventElapsedTime (&ms, begin, end);
  printf("cuda update scaling time = %fms\n",ms);
}


void cuda_get_slices(sp_3matrix * model, real * d_model, real * d_slices, real * d_rot, 
		     real * d_x_coordinates,
		     real * d_y_coordinates, real * d_z_coordinates, int N_slices){
  
  int rows = sp_3matrix_x(model);
  int cols = sp_3matrix_y(model);
  int N_2d = sp_3matrix_x(model)*sp_3matrix_y(model);
  int nblocks = N_slices;
  int nthreads = 256;
  get_slices_kernel<<<nblocks,nthreads>>>(d_model, d_slices, d_rot,d_x_coordinates,
					  d_y_coordinates,d_z_coordinates,
					  rows,cols,
					  sp_3matrix_x(model),sp_3matrix_y(model),
					  sp_3matrix_z(model));
}

real cuda_update_slices(real * d_images, real * d_slices, int * d_mask,
			real * d_respons, real * d_scaling, int N_images, int N_slices, int N_2d,
			sp_3matrix * model, real * d_model,
			real *d_x_coordinates, real *d_y_coordinates,
			real *d_z_coordinates, real *d_rot, real * weights,
			real * d_weight, Setup setup, sp_matrix ** images){
  hipEvent_t begin;
  hipEvent_t end;
  hipEventCreate(&begin);
  hipEventCreate(&end);
  hipEventRecord (begin,0);
  dim3 nblocks = N_slices;
  int nthreads = 256;
  real * d_slices_total_respons;
  hipMalloc(&d_slices_total_respons,sizeof(real)*N_slices);

  real * d_weights;
  hipMalloc(&d_weights,sizeof(real)*N_slices);
  hipMemcpy(d_weights,weights,sizeof(real)*N_slices,hipMemcpyHostToDevice);



  hipEvent_t k_begin;
  hipEvent_t k_end;
  hipEventCreate(&k_begin);
  hipEventCreate(&k_end);
  hipEventRecord (k_begin,0);

  update_slices_kernel<<<nblocks,nthreads>>>(d_images, d_slices, d_mask, d_respons,
					     d_scaling, N_images, N_slices, N_2d,
					     d_slices_total_respons, d_rot,d_x_coordinates,
					     d_y_coordinates,d_z_coordinates,d_model, d_weight,
					     sp_matrix_rows(images[0]),sp_matrix_cols(images[0]),
					     sp_3matrix_x(model),sp_3matrix_y(model),
					     sp_3matrix_z(model),d_weights);  
  hipDeviceSynchronize();
  insert_slices_kernel<<<nblocks,nthreads>>>(d_images, d_slices, d_mask, d_respons,
					     d_scaling, N_images, N_slices, N_2d,
					     d_slices_total_respons, d_rot,d_x_coordinates,
					     d_y_coordinates,d_z_coordinates,d_model, d_weight,
					     sp_matrix_rows(images[0]),sp_matrix_cols(images[0]),
					     sp_3matrix_x(model),sp_3matrix_y(model),
					     sp_3matrix_z(model),d_weights);  
  hipEventRecord(k_end,0);
  hipEventSynchronize(k_end);
  real k_ms;
  hipEventElapsedTime (&k_ms, k_begin, k_end);
  printf("cuda kernel slice update time = %fms\n",k_ms);

  hipError_t status = hipGetLastError();
  if(status != hipSuccess){
    printf("CUDA Error: %s\n",hipGetErrorString(status));
  }
  real slices_total_respons[N_slices];
  hipMemcpy(slices_total_respons,d_slices_total_respons,sizeof(real)*N_slices,
	     hipMemcpyDeviceToHost);
  real overal_respons = 0.0;
  for (int i_slice = 0; i_slice < N_slices; i_slice++) {
    overal_respons += slices_total_respons[i_slice];
  }
  hipMemcpy(model->data,d_model,sizeof(real)*sp_3matrix_size(model),hipMemcpyDeviceToHost);

  hipFree(d_slices_total_respons);
  hipFree(d_weights);
  hipEventRecord(end,0);
  hipEventSynchronize (end);
  real ms;
  hipEventElapsedTime (&ms, begin, end);
  printf("cuda slice update time = %fms\n",ms);
  return overal_respons;
}

real cuda_model_max(real * model, int model_size){
  thrust::device_ptr<real> p(model);
  real max = thrust::reduce(p, p+model_size, real(0), thrust::maximum<real>());
  return max;
}

void cuda_allocate_slices(real ** slices,Setup setup,int N_slices){
  hipMalloc(slices,sizeof(real)*setup.side*setup.side*N_slices);  
}

void cuda_allocate_model(real ** d_model, sp_3matrix * model){
  hipMalloc(d_model,sizeof(real)*sp_3matrix_size(model));
  hipMemcpy(*d_model,model->data,sizeof(real)*sp_3matrix_size(model),hipMemcpyHostToDevice);
}

void cuda_allocate_mask(int ** d_mask, sp_imatrix * mask){
  hipMalloc(d_mask,sizeof(int)*sp_imatrix_size(mask));
  hipMemcpy(*d_mask,mask->data,sizeof(int)*sp_imatrix_size(mask),hipMemcpyHostToDevice);
}

void cuda_allocate_rotations(real ** d_rotations, Quaternion ** rotations,  int N_slices){

  hipMalloc(d_rotations,sizeof(real)*4*N_slices);
  for(int i = 0;i<N_slices;i++){
    hipMemcpy(&(*d_rotations)[4*i],rotations[i]->q,sizeof(real)*4,hipMemcpyHostToDevice);
  }
}

void cuda_allocate_images(real ** d_images, sp_matrix ** images,  int N_images){

  hipMalloc(d_images,sizeof(real)*sp_matrix_size(images[0])*N_images);
  for(int i = 0;i<N_images;i++){
    hipMemcpy(&(*d_images)[sp_matrix_size(images[0])*i],images[i]->data,sizeof(real)*sp_matrix_size(images[0]),hipMemcpyHostToDevice);
  }
}

void cuda_allocate_coords(real ** d_x, real ** d_y, real ** d_z, sp_matrix * x,
			  sp_matrix * y, sp_matrix * z){
  hipMalloc(d_x,sizeof(real)*sp_matrix_size(x));
  hipMalloc(d_y,sizeof(real)*sp_matrix_size(x));
  hipMalloc(d_z,sizeof(real)*sp_matrix_size(x));
  hipMemcpy(*d_x,x->data,sizeof(real)*sp_matrix_size(x),hipMemcpyHostToDevice);
  hipMemcpy(*d_y,y->data,sizeof(real)*sp_matrix_size(x),hipMemcpyHostToDevice);
  hipMemcpy(*d_z,z->data,sizeof(real)*sp_matrix_size(x),hipMemcpyHostToDevice);
}

void cuda_reset_model(sp_3matrix * model, real * d_model){
  hipMemset(d_model,0,sizeof(real)*sp_3matrix_size(model));
}

__global__ void cuda_normalize_model_kernel(real * model, real * weight, int n){
  int i = threadIdx.x + blockIdx.x*blockDim.x;
  if(weight[i] > 0.0f){
    model[i] /= weight[i];
  }else{
    model[i] = 0.0f;
  }
}
void cuda_normalize_model(sp_3matrix * model, real * d_model, real * d_weight){
  int n = sp_3matrix_size(model);
  int nthreads = 256;
  int nblocks = (n+nthreads-1)/nthreads;
  cuda_normalize_model_kernel<<<nblocks,nthreads>>>(d_model,d_weight,n);
  hipDeviceSynchronize();
  thrust::device_ptr<real> p(d_model);
  real model_sum = thrust::reduce(p, p+n, real(0), thrust::plus<real>());
  model_sum /= n;
  /* model /= model_sum; */
  thrust::transform(p, p+n,thrust::make_constant_iterator(1.0f/model_sum), p, thrust::multiplies<real>()); 
}

void cuda_allocate_real(real ** x, int n){
  hipMalloc(x,n);
}
			  
void cuda_allocate_scaling(real ** d_scaling, int N_images){
  hipMalloc(d_scaling,N_images*sizeof(real));
  thrust::device_ptr<real> p(*d_scaling);
  thrust::fill(p, p+N_images, real(1));
}

__global__ void cuda_normalize_responsabilities_kernel(real * respons, int N_slices, int N_images){
  __shared__ real cache[256];
  int i_image = blockIdx.x;
  int tid = threadIdx.x;
  int step = blockDim.x;
  cache[tid] = -1.0e10f;
  for(int i_slice = tid;i_slice < N_slices;i_slice += step){
    if(cache[tid] < respons[i_slice*N_images+i_image]){
      cache[tid] = respons[i_slice*N_images+i_image];
    }
  }
  inblock_maximum(cache);
  real max_resp = cache[0];
  for (int i_slice = tid; i_slice < N_slices; i_slice+= step) {
    respons[i_slice*N_images+i_image] -= max_resp;
  }
  cache[tid] = 0;
  for (int i_slice = tid; i_slice < N_slices; i_slice+=step) {
    if (respons[i_slice*N_images+i_image] > -1.0e10f) {
      respons[i_slice*N_images+i_image] = expf(respons[i_slice*N_images+i_image]);
      cache[tid] += respons[i_slice*N_images+i_image];
    } else {
      respons[i_slice*N_images+i_image] = 0.0f;
    }
  }
  inblock_reduce(cache);
  real sum = cache[0];
  for (int i_slice = tid; i_slice < N_slices; i_slice+=step) {
    respons[i_slice*N_images+i_image] /= sum;
  }
  
}

void cuda_normalize_responsabilities(real * d_respons, int N_slices, int N_images){
  int nblocks = N_images;
  int nthreads = 256;
  cuda_normalize_responsabilities_kernel<<<nblocks,nthreads>>>(d_respons,N_slices,N_images);
}

// x_log_x<T> computes the f(x) -> x*log(x)
template <typename T>
struct x_log_x
{
  __host__ __device__
  T operator()(const T& x) const { 
    if(x > 0){
      return x * logf(x);
    }else{
      return 0;
    }
  }
};

real cuda_total_respons(real * d_respons, real * respons,int n){
  thrust::device_ptr<real> p(d_respons);
  x_log_x<real> unary_op;
  thrust::plus<real> binary_op;
  real init = 0;
  // Calculates sum_0^n d_respons*log(d_respons)
  return thrust::transform_reduce(p, p+n, unary_op, init, binary_op);
}